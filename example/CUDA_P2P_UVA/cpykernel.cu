#include <hip/hip_runtime.h>
#include <iostream>

// CUDA核函数，用于复制数据
__global__ void copyKernel(float* dest, const float* src, size_t numElements) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numElements) {
        dest[idx] = src[idx];
    }
}

// 初始化为0的CUDA内核
__global__ void initZero(float* ptr, size_t size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        ptr[idx] = 0.0f;
    }
}

// 初始化为1的CUDA内核
__global__ void initOne(float* ptr, size_t size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        ptr[idx] = 1.0f;
    }
}

// 用于检查CUDA调用的状态
void checkCudaStatus(hipError_t status) {
    if (status != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(status) << std::endl;
        exit(1);
    }
}

int main() {
    const size_t Nx = 1024;
    const size_t Ny = 1024;
    const size_t Nz = 1024;
    const size_t totalElements = Nx * Ny * Nz;
    const size_t size = totalElements * sizeof(float);
    
    float *A, *B;
    hipEvent_t start, stop;
    float milliseconds = 0;

    // 创建事件
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // 在GPU 0上分配并初始化数组A
    hipSetDevice(0);
    checkCudaStatus(hipMalloc(&A, size));
    initZero<<<(totalElements + 255) / 256, 256>>>(A, totalElements);
    checkCudaStatus(hipGetLastError());
    checkCudaStatus(hipDeviceSynchronize());

    // 在GPU 1上分配并初始化数组B
    hipSetDevice(1);
    checkCudaStatus(hipMalloc(&B, size));
    initOne<<<(totalElements + 255) / 256, 256>>>(B, totalElements);
    checkCudaStatus(hipGetLastError());
    checkCudaStatus(hipDeviceSynchronize());

    // 启用P2P访问
    hipSetDevice(0);
    checkCudaStatus(hipDeviceEnablePeerAccess(1, 0));

    // 记录复制开始的时间
    hipEventRecord(start);

    // 执行复制核函数
    int threadsPerBlock = 256;
    int blocksPerGrid = (totalElements + threadsPerBlock - 1) / threadsPerBlock;
    copyKernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, totalElements);
    checkCudaStatus(hipGetLastError());
    checkCudaStatus(hipDeviceSynchronize());

    // 记录复制结束的时间并等待事件完成
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // 计算并打印所用时间
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Time for manual P2P copy: " << milliseconds / 1000.0f << " seconds." << std::endl;

    // 清理
    hipSetDevice(0);
    hipFree(A);
    hipSetDevice(1);
    hipFree(B);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
